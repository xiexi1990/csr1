#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <cnpy.h>

using namespace std;

struct SparseMatrix {
    int n; // 矩阵维度
    vector<int> row_ptr; // 行指针
    vector<int> col_idx; // 列索引
    vector<float> values; // 非零元素

    SparseMatrix(int n) {
        this->n = n;
        row_ptr.resize(n+1);
        row_ptr[0] = 0;
    }
};

SparseMatrix generate_random_sparse_matrix(int n, int num_nonzeros) {
    SparseMatrix mat(n);

    int max_nonzeros_per_row = num_nonzeros / n;
    int remaining_nonzeros = num_nonzeros - max_nonzeros_per_row * n;

    srand(123);

    int k = 0;
    for (int i = 0; i < n; i++) {
        int num_nonzeros_in_row = max_nonzeros_per_row;
        if (remaining_nonzeros > 0) {
            num_nonzeros_in_row += rand() % 2;
            remaining_nonzeros--;
        }
        for (int j = 0; j < num_nonzeros_in_row; j++) {
            float val = (float) rand() / RAND_MAX; // 随机生成一个0到1之间的数
            int col = rand() % n; // 随机选择一个列
            mat.col_idx.push_back(col);
            mat.values.push_back(val);
            k++;
        }
        mat.row_ptr[i+1] = k;
    }

    return mat;
}

int main() {
    int n = 50; // 矩阵维度
    int num_nonzeros = 300; // 矩阵非零元素个数
    int ndim = 32;

    SparseMatrix mat = generate_random_sparse_matrix(n, num_nonzeros);


    cnpy::npz_t npz = cnpy::npz_load("npz36.npz");
    cnpy::NpyArray npy_shape = npz["shape"];
    uint32_t num_rows = npy_shape.data<uint32_t>()[0];
    uint32_t num_cols = npy_shape.data<uint32_t>()[2];
    cnpy::NpyArray npy_data = npz["data"];
    uint32_t nnz = npy_data.shape[0];
    cnpy::NpyArray npy_indices = npz["indices"];
    cnpy::NpyArray npy_indptr = npz["indptr"];
    // csr_matrix.adj_data.insert(csr_matrix.adj_data.begin(), &npy_data.data<float>()[0],
    //     &npy_data.data<float>()[nnz]);
    // csr_matrix.adj_indices.insert(csr_matrix.adj_indices.begin(), &npy_indices.data<uint32_t>()[0],
    //     &npy_indices.data<uint32_t>()[nnz]);
    // csr_matrix.adj_indptr.insert(csr_matrix.adj_indptr.begin(), &npy_indptr.data<uint32_t>()[0],
    //     &npy_indptr.data<uint32_t>()[num_rows + 1]);


    int *ptr, *idx;
    float *val, *vin, *vout;
    hipMallocManaged(&ptr, (n+1) * sizeof(int));
    hipMallocManaged(&idx, num_nonzeros * sizeof(int));
    hipMallocManaged(&val, num_nonzeros * sizeof(float));
    hipMallocManaged(&vin, n * ndim * sizeof(float));
    hipMallocManaged(&vout, n * ndim * sizeof(float));

    memset(vout, 0, n * ndim * sizeof(float));
    memcpy(ptr, mat.row_ptr.data(), (n+1) * sizeof(int));


    hipFree(ptr);
    hipFree(idx);
    hipFree(val);
    hipFree(vin);
    hipFree(vout);

    return 0;
}
